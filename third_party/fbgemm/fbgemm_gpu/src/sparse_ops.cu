#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "fbgemm_gpu/quantize_ops.cuh"
#include "fbgemm_gpu/sparse_ops.cuh"
#include "fbgemm_gpu/sparse_ops.h"
#include "fbgemm_gpu/sparse_ops_utils.h"

#include <ATen/ATen.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/library.h>

#include "ATen/Parallel.h"
#include "cub/device/device_scan.cuh"

namespace at {

Tensor asynchronous_exclusive_cumsum(const Tensor& t_in) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(t_in.get_device());
  size_t temp_storage_bytes = 0;
  TORCH_CHECK(t_in.is_contiguous());
  TORCH_CHECK(t_in.dtype() == kInt || t_in.dtype() == kLong);
  // CUB only handles up to INT_MAX elements.
  TORCH_CHECK(t_in.numel() < std::numeric_limits<int32_t>::max());
  auto t_out = at::empty_like(t_in);
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper1", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  auto temp_storage = at::empty(
      {static_cast<int64_t>(temp_storage_bytes)}, t_in.options().dtype(kByte));
  AT_DISPATCH_INTEGRAL_TYPES(
      t_in.scalar_type(), "cub_exclusive_sum_wrapper2", ([&] {
        AT_CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(
            temp_storage.data_ptr(),
            temp_storage_bytes,
            t_in.data_ptr<scalar_t>(),
            t_out.data_ptr<scalar_t>(),
            t_in.numel(),
            at::cuda::getCurrentCUDAStream()));
      }));
  return t_out;
}

std::tuple<Tensor, Tensor, c10::optional<Tensor>> permute_sparse_data_cuda(
    const Tensor& permute,
    const Tensor& lengths,
    const Tensor& indices,
    const c10::optional<Tensor>& weights) {
  TENSOR_ON_CUDA_GPU(permute);
  TENSOR_ON_CUDA_GPU(lengths);
  TENSOR_ON_CUDA_GPU(indices);
  TENSOR_ON_CUDA_GPU(weights);

  TENSORS_ON_SAME_DEVICE(permute, lengths);
  TENSORS_ON_SAME_DEVICE(permute, indices);
  TENSORS_ON_SAME_DEVICE(permute, weights);

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(indices.get_device());

  const auto permute_contig = permute.contiguous();
  const auto lengths_contig = lengths.contiguous();
  const auto indices_contig = indices.contiguous();
  // the data to permute over can be less or more with or without
  // repetitions
  const auto T = permute.numel();
  const auto T_ = lengths.size(0);
  const auto B = lengths.view({lengths.sizes()[0], -1}).sizes()[1];

  Tensor permuted_lengths;
  Tensor permuted_indices;
  Tensor permuted_weights;

  permuted_lengths = at::empty({T, B}, lengths.options());

  constexpr int32_t threads_1 = 256;
  const auto blocks_1 = cuda_calc_xblock_count(B * T, threads_1);
  AT_DISPATCH_INDEX_TYPES(
      lengths.scalar_type(), "permute_lengths_kernel", ([&] {
        permute_lengths_kernel<index_t>
            <<<blocks_1, threads_1, 0, at::cuda::getCurrentCUDAStream()>>>(
                T,
                B,
                lengths_contig.data_ptr<index_t>(),
                permute.data_ptr<int32_t>(),
                permuted_lengths.data_ptr<index_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));

  // convert lengths to offsets
  const auto input_offsets = asynchronous_exclusive_cumsum(lengths_contig);
  const auto output_offsets = asynchronous_exclusive_cumsum(permuted_lengths);
  int64_t permuted_lengths_sum = indices.numel();

  /* TODO: Remove the condition protecting the slow path because even when the
   * condition below is true permuted_lengths.sum() could still be needed. For
   * instance if there are three features with indices `[0, 1, 2]`, `permute`
   * can be `[0, 1, 1]` for which permuted lengths sum would be needed to create
   * permuted_{indices, weights} and `permuted_lengths_sum = indices.numel() or
   * weights.numdel() would be incorrect.
   */
  if (T_ != T) {
    permuted_lengths_sum = permuted_lengths.sum().item<int64_t>();
  }

  constexpr int32_t BT_blocks = 32;
  dim3 threads_2(32, BT_blocks);
  const auto blocks_2 = cuda_calc_xblock_count(B * T, BT_blocks);
  permuted_indices = at::empty(permuted_lengths_sum, indices.options());

  AT_DISPATCH_INDEX_TYPES(
      input_offsets.scalar_type(), "permute_data_kernel_1", ([&] {
        using offsets_t = index_t;
        AT_DISPATCH_ALL_TYPES(
            indices.scalar_type(), "permute_data_kernel_2", ([&] {
              using indices_t = scalar_t;
              if (weights.has_value()) {
                const Tensor weights_value = weights.value();
                const auto weights_value_contig = weights_value.contiguous();
                permuted_weights =
                    at::empty(permuted_lengths_sum, weights_value.options());
                AT_DISPATCH_FLOATING_TYPES(
                    weights_value.scalar_type(), "permute_data_kernel_3", ([&] {
                      using weights_t = scalar_t;
                      permute_data_kernel<true, offsets_t, indices_t, weights_t>
                          <<<blocks_2,
                             threads_2,
                             0,
                             at::cuda::getCurrentCUDAStream()>>>(
                              permuted_lengths_sum,
                              T,
                              B,
                              indices_contig.data_ptr<indices_t>(),
                              weights_value_contig.data_ptr<weights_t>(),
                              permute_contig.data_ptr<int32_t>(),
                              input_offsets.data_ptr<offsets_t>(),
                              output_offsets.data_ptr<offsets_t>(),
                              permuted_indices.data_ptr<indices_t>(),
                              permuted_weights.data_ptr<weights_t>());
                      C10_CUDA_KERNEL_LAUNCH_CHECK();
                    })); // for each weights_t
              } else {
                permute_data_kernel<false, offsets_t, indices_t, std::nullptr_t>
                    <<<blocks_2,
                       threads_2,
                       0,
                       at::cuda::getCurrentCUDAStream()>>>(
                        permuted_lengths_sum,
                        T,
                        B,
                        indices_contig.data_ptr<indices_t>(),
                        nullptr,
                        permute_contig.data_ptr<int32_t>(),
                        input_offsets.data_ptr<offsets_t>(),
                        output_offsets.data_ptr<offsets_t>(),
                        permuted_indices.data_ptr<indices_t>(),
                        nullptr);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
            })); // for each indices_t
      })); // for each offsets_t
  return {permuted_lengths, permuted_indices, permuted_weights};
}

at::Tensor _float_to_fused8bitrowwise_gpu(const at::Tensor& input) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned + 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  auto output = at::empty(
      output_dims, // 4 = sizeof(float)
      input.options().dtype(at::kByte));

  if (nrows == 0 || ncols == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;
  const auto num_blocks = cuda_calc_xblock_count(nrows, threads_per_block);
  // think unsigned as we use 0, 255

  if (nrows <= 20) {
    _float_to_fused8bitrowwise_cuda_kernel<<<
        num_blocks,
        threads_per_block,
        0,
        at::cuda::getCurrentCUDAStream()>>>(
        input.data_ptr<float>(), nrows, ncols, output.data_ptr<std::uint8_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    // range_tensor is used to store the range for each embedding row.
    // We save range/255.0f as row scale, and use 255.0f / (range + kEpsilon) to
    // quantize. This will guarantee the numerical match but bring some perf
    // regression.
    auto range_tensor = at::empty({nrows}, input.options().dtype(at::kFloat));

    {
      // we need a blockDim.x that is a power of 2 no larger than the warp size
      // of 32

      int blockDim_x = 1;
      if (ncols > 16) {
        // max warp size
        blockDim_x = 32;
      } else {
        while (blockDim_x < ncols) {
          blockDim_x <<= 1;
        }
      }

      const int rows_per_block = threads_per_block / blockDim_x;
      const auto num_blocks_warp =
          cuda_calc_xblock_count(nrows, rows_per_block);

      _get_8bit_qparam_cuda_kernel<<<
          num_blocks_warp,
          dim3(blockDim_x, rows_per_block),
          0,
          at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<float>(),
          nrows,
          ncols,
          output.data_ptr<std::uint8_t>(),
          range_tensor.data_ptr<float>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }

    {
      const int blockDim_x = std::min(ncols, threads_per_block);
      dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);
      const auto gridDim_x = cuda_calc_xblock_count(ncols, blockDim.x);
      const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
      dim3 gridDim(gridDim_x, gridDim_y);

      _compute_8bit_quantize_cuda_kernel<<<
          gridDim,
          blockDim,
          0,
          at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<float>(),
          range_tensor.data_ptr<float>(),
          nrows,
          ncols,
          output.data_ptr<std::uint8_t>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }

  return output;
}

at::Tensor _fused8bitrowwise_to_float_gpu(const at::Tensor& input) {
  TENSOR_ON_CUDA_GPU(input);
  TORCH_CHECK(input.is_contiguous(), "input must be contiguous");

  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(input.get_device());

  const auto input_sizes = input.sizes();
  const auto last_dim = input_sizes.size() - 1;
  const int nrows = c10::size_to_dim_(last_dim, input_sizes);
  const int ncols = input_sizes[last_dim];
  const int ncols_aligned = (ncols + 4 - 1) / 4 * 4;
  const int output_columns = ncols_aligned - 2 * sizeof(float);

  // Global memory instructions support reading or writing words of size equal
  // to 1, 2, 4, 8, or 16 bytes. Any access (via a variable or a pointer) to
  // data residing in global memory compiles to a single global memory
  // instruction if and only if the size of the data type is 1, 2, 4, 8, or 16
  // bytes and the data is naturally aligned (i.e., its address is a multiple of
  // that size).
  auto output_dims = input_sizes.vec();
  output_dims[last_dim] = output_columns;
  auto output = at::empty(
      output_dims, // 4 = sizeof(float)
      input.options().dtype(at::kFloat));

  if (nrows == 0 || output_columns == 0) {
    return output;
  }

  constexpr int threads_per_block = 256;

  const int blockDim_x = std::min(threads_per_block, output_columns);
  dim3 blockDim(blockDim_x, threads_per_block / blockDim_x);

  const auto gridDim_x = cuda_calc_xblock_count(output_columns, blockDim.x);
  const auto gridDim_y = cuda_calc_block_count(nrows, blockDim.y);
  dim3 gridDim(gridDim_x, gridDim_y);

  _fused8bitrowwise_to_float_cuda_kernel<<<
      gridDim,
      blockDim,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      input.data_ptr<std::uint8_t>(), nrows, ncols, output.data_ptr<float>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

} // namespace at
